#include "hip/hip_runtime.h"
// All C includes must be wrapped in extern "C"
extern "C" {
#include "global.h"
#include "util.h"
#include "constants.h"
}
#include <assert.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>

#define PIXEL_SIZE	(128)
#define PIXEL_BLOCK	(8)
#define TILE_SIZE (4)
#define THREAD_ITER	(1)
#define GRID_SIZE (PIXEL_SIZE/TILE_SIZE)

float *lens_x;
float *lens_y;
float *lens_mass;
size_t nobjects;

/* Pointers to the lens x,y co-ordinates on the GPU device */
float *d_lens_x;
float *d_lens_y;
float *d_lens_mass;

typedef struct vars {
  unsigned int rpp;
  float kappa_c, gamma_, source_scale;
  float image_scale_x, image_scale_y;
  float increment_x, increment_y;
} vars;

void init_var(vars *var) {
  var->rpp = 1;
  var->kappa_c = kappa_c;
  var->gamma_ = gamma_;
  var->source_scale = source_scale;
  var->image_scale_x = image_scale_x;
  var->image_scale_y = image_scale_y;
  var->increment_x = 0;
  var->increment_y = 0;
}

int highest(unsigned int *results, unsigned int size) {
  unsigned int i, highest_count = 0;
  for(i = 0; i < size; ++i){
    if (results[i] > highest_count)
      highest_count = results[i];
  }
  return highest_count;
}

int total_r(unsigned int *results, unsigned int size){
  unsigned int i, total = 0;
  for(i = 0; i < size; ++i){
        total += results[i];
  }
  return total;
}

__global__ void glensing(const float *lens_x, const float *lens_y, const float *lens_mass, const float pixelBlock, 
					const size_t nobjects, unsigned int* results, const vars* v, hiprandState_t *state) {
	
  	const int col = blockDim.x * blockIdx.x;
	const int row = pixelBlock * PIXEL_BLOCK + blockDim.y * blockIdx.y;
/* 
	const float base_x = (-v->image_scale_x) + row*v->increment_x;
  	const float base_y = (-v->image_scale_y) + col*v->increment_y;	
  	
	//Position of each light ray inside each Block
	const int bx = threadIdx.x;
	const int by = threadIdx.y;  
	
	const float unit_x = v->increment_x/TILE_SIZE;
	const float unit_y = v->increment_y/TILE_SIZE;
	
	const float source_scale = v->source_scale;
	
	float start_x, start_y, dx, dy;
	size_t k;
	float dist;
		    
    start_x = base_x + bx * unit_x ;
    start_y = base_y + by * unit_y;

    dx = (1-v->gamma_)*start_x - v->kappa_c*start_x;
    dy = (1+v->gamma_)*start_y - v->kappa_c*start_y;

    for(k = 0; k < nobjects; ++k) {
      dist = pow(start_x - lens_x[k], 2) + pow(start_y - lens_y[k], 2);
      dx -= lens_mass[k] * (start_x - lens_x[k]) / dist;
      dy -= lens_mass[k] * (start_y - lens_y[k]) / dist;
    }
    	
    if ((dx >= -source_scale/2) && (dx <= source_scale/2) &&
        (dy >= -source_scale/2) && (dy <= source_scale/2)) {
     	 
     	 int px = (dx + source_scale/2) / (source_scale/PIXEL_SIZE);
     	 int py = PIXEL_SIZE - (dy + source_scale/2) / (source_scale/PIXEL_SIZE);
    	 
    	 atomicAdd(&results[py * PIXEL_SIZE + px], 1);
    }*/
    atomicAdd(&results[row * PIXEL_SIZE + col], 1);
	
}

int main(int argc, char** argv) {  
  float increment_x, increment_y;
  // Load relevant settings and data
  if (argc < 2) error("Requires argument with lens positions and optional mass");
  setup_constants();
  vars *variables = (vars *)salloc(sizeof(vars));
  init_var(variables);
  read_lenses(argv[1]);

  fprintf(stderr, "X %f and Y %f\n", image_scale_x, image_scale_y);
  increment_x = (image_scale_x * 2) / PIXEL_SIZE;
  increment_y = (image_scale_y * 2) / PIXEL_SIZE;
  variables->increment_x = increment_x;
  variables->increment_y = increment_y;
  fprintf(stderr, "Increments for X %f and Y %f\n", increment_x, increment_y);

  unsigned int *results = (unsigned int *)calloc(PIXEL_SIZE * PIXEL_SIZE, sizeof(unsigned int));
  unsigned int *d_results;
  if (!results) error("calloc failed in allocating the result array");

  // Setting up CUDA global memory
  vars *d_variables;
  hiprandState_t *state;
  hipMalloc(&state, sizeof(hiprandState_t));
  hipMalloc(&d_lens_x, sizeof(float) * nobjects);
  hipMalloc(&d_lens_y, sizeof(float) * nobjects);
  hipMalloc(&d_lens_mass, sizeof(float) * nobjects);

  hipMalloc(&d_results, PIXEL_SIZE*PIXEL_SIZE*sizeof(unsigned int));
  hipMalloc(&d_variables, sizeof(vars));

  hipMemset(d_results, 0, PIXEL_SIZE*PIXEL_SIZE*sizeof(unsigned int));
  hipMemcpy(d_variables, variables, sizeof(vars), hipMemcpyHostToDevice);
  hipMemcpy(d_lens_x, lens_x, sizeof(float) * nobjects, hipMemcpyHostToDevice);
  hipMemcpy(d_lens_y, lens_y, sizeof(float) * nobjects, hipMemcpyHostToDevice);
  hipMemcpy(d_lens_mass, lens_mass, sizeof(float) * nobjects, hipMemcpyHostToDevice);


  // Perform gravitational microlensing
  dim3 gdim(PIXEL_SIZE, PIXEL_BLOCK);
  dim3 bdim(TILE_SIZE, TILE_SIZE);
  int proc_row;
  
  for(proc_row=0; proc_row<PIXEL_SIZE/PIXEL_BLOCK; ++proc_row){
	  glensing<<<gdim, bdim>>>(d_lens_x, d_lens_y, d_lens_mass, proc_row, nobjects, d_results, d_variables, state);
	  hipDeviceSynchronize();  
  }
  
  hipMemcpy(results, d_results, PIXEL_SIZE*PIXEL_SIZE*sizeof(unsigned int), hipMemcpyDeviceToHost);

  int highest_c = highest(results, PIXEL_SIZE * PIXEL_SIZE);
  int total = total_r(results, PIXEL_SIZE * PIXEL_SIZE);
  write_pgm(results, PIXEL_SIZE, PIXEL_SIZE, highest_c);
  printf("the number of total rays is %d\n", total);
  // Free the memory allocated during processing
  // GPU
  hipFree(d_lens_x);
  hipFree(d_lens_y);
  hipFree(d_lens_mass);
  hipFree(d_results);
  hipFree(d_variables);
  hipFree(state);
  // CPU
  free(lens_x);
  free(lens_y);
  free(lens_mass);
  free(results);

  return 0;
}
