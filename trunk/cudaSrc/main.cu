#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "utils.h"
#include "constants.h"

#define BLOCK_SIZE	16
#define PIXEL_SIZE	512


#define  

float *lens_x;
float *lens_y;
float *lens_mass;
size_t nobjects;

/*device lens_x and lens_y*/
float *d_lens_x;
float *d_lens_y;

/* read_lenses • Loads a lens file of format {x, y, (optional)mass} and allocate the correct sized array for the attributes */
void read_lenses(const char *filename) {
  size_t i, len = 0;
  float x, y;
  char c, *tmp, *line = NULL;
  FILE *fp;

  fprintf(stderr, "Reading in lenses...\n");
  if (!(fp = fopen(filename, "r"))) error("Can't open lens file...");

  nobjects = 0;
  // Count the number of lenses we must allocate for (one per line)
  while ((c = getc(fp)) != EOF) {
    if (c == '\n') ++nobjects;
  }
  fprintf(stderr, "Total lenses found: %d\n", nobjects);
  // Seek to the start of the file for actual reading
  fseek(fp, 0, SEEK_SET);

  // Allocate memory for the lenses
  lens_x = (float *)salloc(sizeof(float) * nobjects);
  lens_y = (float *)salloc(sizeof(float) * nobjects);
  lens_mass = (float *)salloc(sizeof(float) * nobjects);

  for(i = 0; i < nobjects; ++i) {
    if(fscanf(fp, "%f %f", &x, &y)!=2) error("wrong data!\n");
    lens_x[i] = x;
    lens_y[i] = y;
    lens_mass[i] = 1;
    
  }

  if (fclose(fp) != 0) error("Can't close lens file...");
  // Deallocate memory used by line
  free(line);
} 

int highest(int *result, int size){
	int i = 0, highest_count=0;
	for(; i<size; ++i){
		if (results[i] > highest_count) 
			highest_count = results[i];
	}
	return highest_count;
}

/* write_pgm • Output the results as a PGM (portable gray map) image for review */
void write_pgm(int *results, int pixel_x, int pixel_y, int highest) {
	FILE *fout;
	fprintf(stderr, "Writing resulting image...\n");
	if (!(fout = fopen("img.pgm", "w"))) error("Can't open results file...");
	// Writing the PGM format which starts with P2
	fprintf(fout, "P2\n");
	// Followed by pixel width, height and the value considered white
	fprintf(fout, "%d %d\n", pixel_x, pixel_y);
	fprintf(fout, "%d\n", highest);
	// Print each value in a row of WIDTH length
	int px, py;
	for(py = 0; py < pixel_y; ++py) {
	  for(px = 0; px < pixel_x; ++px) {
	    fprintf(fout, "%d ", results[py * pixel_x + px]);
	  }
	  fprintf(fout, "\n");
	}
	if (fclose(fout) != 0) error("Can't close results file...");
}


__global__ void kernel(const float *lens_x, const float *lens_y, int* result, const int iter){
  	const int x = blockIdx.x; 
  	const int y = iter;
  	const int i = threadIdx.x;
  	const int j = threadIdx.y;
  	int dx = x, dy = y;
  	
  	/* deflection calculation*/
  
  	
  	/*finish calculation*/
  	
  	result[dy*PIXEL_SIZE + dx] ++;
}


int main(int argc, char** argv)
{
	float x, y, dx, dy, increment_x, increment_y;
	int pixel_x = 512, pixel_y = 512, it;
	
	// Load relevant settings and data
	if (argc < 2) error("Requires argument with lens positions and optional mass");
	setup_constants();
	read_lenses(argv[1]);
	fprintf(stderr, "X %f and Y %f\n", image_scale_x, image_scale_y);
	increment_x = (image_scale_x * 2) / (pixel_x*10);
	increment_y = (image_scale_y * 2) / (pixel_y*10);
	fprintf(stderr, "Increments for X %f and Y %f\n", increment_x, increment_y);
	
	int *results = (int *)calloc(pixel_x * pixel_y, sizeof(float));
	int *d_results;
	if (!results) error("calloc failed in allocating the result array");
  
    /* here starts the cuda*/
    hipMalloc(&d_lens_x, sizeof(float) * nobjects);
 	hipMalloc(&d_lens_y, sizeof(float) * nobjects);
 	hipMalloc(&d_results, PIXEL_SIZE*PIXEL_SIZE*sizeof(float));
 	
 	hipMemset(d_results, 0, PIXEL_SIZE*PIXEL_SIZE*sizeof(float));
  	
  	hipMemcpy(d_lens_x, lens_x, sizeof(float) * nobjects, hipMemcpyHostToDevice);
  	hipMemcpy(d_lens_y, lens_y, sizeof(float) * nobjects, hipMemcpyHostToDevice);
      
  	dim3 dimb(BLOCK_SIZE, BLOCK_SIZE);
  	dim3 dimg(PIXEL_SIZE);
  	
  	for(int i=0; i<=PIXEL_SIZE; ++i){
  		kernel<<<dimg, dimb>>>(d_lens_x, d_lens_y, d_results, i);
  		fprintf(stderr, "\r%1.0f%% ", 100*i*1.0/PIXEL_SIZE);
  	}	
  	
  	hipMemcpy(d_results, results,PIXEL_SIZE*PIXEL_SIZE*sizeof(float), hipMemcpyDeviceToHost);

  	hipFree(d_lens_x);
 	hipFree(d_lens_y);
	hipFree(d_results);

  	
  	/* cuda kernel finishes*/
  	fprintf(stderr, "\n");
 	write_pgm(results, pixel_x, pixel_y, highest(results, PIXEL_SIZE*PIXEL_SIZE));  	
 	
	return 0;
}
