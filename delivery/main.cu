#include "hip/hip_runtime.h"
#include "global.h"
#include "util.h"
#include "constants.h"
#include "tree_struct.h"
	
#include <assert.h>
#include <stdio.h>
#include <omp.h>
#include <hip/hip_runtime.h>
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"

#define PIXEL_SIZE 	(4096)

/*put the number of GPUs available here*/
#define NUM_DEVICE	(1)
#define KERNEL_CALL_PER_DEVICE	(1)
#define KERNEL_CALL_NUM	(KERNEL_CALL_PER_DEVICE * NUM_DEVICE)

#define TILE_SIZE	(16)
#define GRID_SIZE 	(PIXEL_SIZE/TILE_SIZE)

#define SEED_MODE	1024


//number of rays per pixel
#define RPP (4)
#define	RPP_Kernel	(1)
#define ITERATION	(RPP/(KERNEL_CALL_NUM*RPP_Kernel))

/*Pointers to the lens x,y co-ordinates and mass on the global memory*/
float *lens_x;
float *lens_y;
float *lens_mass;
size_t nobjects;

/*setting up some key parameters for the */
void init_variables(d_constants *const_struct) {
	const_struct->rpp = RPP_Kernel;
	const_struct->kappa_c = kappa_c;
	const_struct->gamma_ = gamma_;
	const_struct->source_scale = source_scale;
	const_struct->image_scale_x = image_scale_x;
	const_struct->image_scale_y = image_scale_y;
	const_struct->increment_x = 0;
	const_struct->increment_y = 0;
}

__global__ void curand_setup(hiprandState* globalState, unsigned long seed){
	const unsigned int row = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int col = blockIdx.y*blockDim.y + threadIdx.y;
	const unsigned int id = row*PIXEL_SIZE + col;
	hiprand_init(seed, id, 0, &globalState[id]);
}

__global__ void glensing(const float *lens_x, const float *lens_y, const float *lens_mass, const size_t nobjects, unsigned int* results, 
							const d_constants* v, hiprandState* globalState, long int seed) {
	const unsigned int row = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int col = blockIdx.y*blockDim.y + threadIdx.y;
	const unsigned int id = blockIdx.x*blockDim.x + blockDim.y;
	const unsigned int position = row*PIXEL_SIZE + col;
	const float initial_x = (-v->image_scale_x) + row*v->increment_x;
	const float initial_y = (-v->image_scale_y) + col*v->increment_y;
	const float increment_x = v->increment_x;
	const float increment_y = v->increment_y;
	const float source_scale = v->source_scale;
	
	float start_x, start_y, dx, dy;
	size_t k, it;
	__device__ __shared__ hiprandState localState;
	localState = globalState[id];
	
	for(it=0; it<RPP_Kernel; ++it){
		start_x = initial_x + hiprand_uniform(&localState) * increment_x;
		start_y = initial_y + hiprand_uniform(&localState) * increment_y;
		globalState[id] = localState;
		dx = (1-v->gamma_)*start_x - v->kappa_c*start_x;
		dy = (1+v->gamma_)*start_y - v->kappa_c*start_y;
	
		for(k = 0; k < nobjects; ++k) {
			float dist = pow(start_x - lens_x[k], 2) + pow(start_y - lens_y[k], 2);
			dx -= lens_mass[k] * (start_x - lens_x[k]) / dist;
			dy -= lens_mass[k] * (start_y - lens_y[k]) / dist;
		}
	
		if ((dx >= -source_scale/2) && (dx <= source_scale/2) &&
			(dy >= -source_scale/2) && (dy <= source_scale/2)) {
			int px = (dx + source_scale/2) / (source_scale/PIXEL_SIZE);
			int py = PIXEL_SIZE - (dy + source_scale/2) / (source_scale/PIXEL_SIZE);
			atomicAdd(&results[py * PIXEL_SIZE + px], 1);
		}
	}
}

int main(int argc, char** argv){	
	
	printf("PIXEL SIZE %d x %d\n", PIXEL_SIZE, PIXEL_SIZE);
	int i;
	float increment_x, increment_y;
	
// Load relevant settings and data
	if (argc < 2) error("Requires argument with lens positions and optional mass");
	setup_constants();
	
	d_constants *const_struct = (d_constants*)salloc(sizeof(d_constants));
	init_variables(const_struct);
	read_lenses(argv[1]);

	printf("X %f and Y %f\n", image_scale_x, image_scale_y);
	increment_x = (image_scale_x * 2) / (PIXEL_SIZE);
	increment_y = (image_scale_y * 2) / (PIXEL_SIZE);
	const_struct->increment_x = increment_x;
	const_struct->increment_y = increment_y;
	printf("Increments for X %f and Y %f\n", increment_x, increment_y);

	int num_devices;
	hipGetDeviceCount (&num_devices); 
	if(num_devices != NUM_DEVICE) error("Wrong configuration on the number of devices, please re-confirm the number of devices");

	printf("---------------------------\n");
	printf("%d GPGPU device found:\n", num_devices);
	for(i = 0; i < num_devices; i++)
	{
		hipDeviceProp_t dprop;
		hipGetDeviceProperties(&dprop, i);
				printf("   %d: %s\n", i, dprop.name);
	}
	printf("---------------------------\n");
	
	unsigned int *results[KERNEL_CALL_NUM];

	for(i=0; i<KERNEL_CALL_NUM; ++i){
		results[i] = (unsigned int *)calloc(PIXEL_SIZE * PIXEL_SIZE, sizeof(unsigned int));
		if (!results[i]) error("calloc failed in allocating the result array");
	}
	
	unsigned int *final_result = (unsigned int *)calloc(PIXEL_SIZE * PIXEL_SIZE, sizeof(unsigned int));

	omp_set_num_threads(KERNEL_CALL_NUM);
	#pragma omp parallel 
	{
		int device_No = omp_get_thread_num()%NUM_DEVICE;
		unsigned int thread_No = omp_get_thread_num();
		hipSetDevice(device_No);
		/* Pointers to the lens x,y co-ordinates and mass on the GPU device */
		float *d_lens_x;
		float *d_lens_y;
		float *d_lens_mass;
		d_constants *d_const_struct;
		hiprandState* globalState;
		int error = 0;
		
		hipMalloc(&d_lens_x, sizeof(float) * nobjects);
		hipMalloc(&d_lens_y, sizeof(float) * nobjects);
		hipMalloc(&d_lens_mass, sizeof(float) * nobjects);	
		hipMalloc(&d_const_struct, sizeof(d_constants));
		hipMalloc(&globalState, sizeof(hiprandState)*GRID_SIZE*GRID_SIZE);
		
		dim3 bdim(1, 1);
		dim3 gdim(GRID_SIZE, GRID_SIZE); 
		curand_setup<<<gdim, bdim>>>(globalState, time(NULL));
		
		hipMemcpy(d_const_struct, const_struct, sizeof(d_constants), hipMemcpyHostToDevice);
		hipMemcpy(d_lens_x, lens_x, sizeof(float) * nobjects, hipMemcpyHostToDevice);
		hipMemcpy(d_lens_y, lens_y, sizeof(float) * nobjects, hipMemcpyHostToDevice);
		hipMemcpy(d_lens_mass, lens_mass, sizeof(float) * nobjects, hipMemcpyHostToDevice);


		int iteration_No = 1;
		
		for(; iteration_No<=ITERATION&&!error; ++iteration_No){
			unsigned int *d_results;
			hipMalloc(&d_results, PIXEL_SIZE*PIXEL_SIZE*sizeof(unsigned int));
			hipMemset(d_results, 0, PIXEL_SIZE*PIXEL_SIZE*sizeof(unsigned int));
			unsigned int *temp_results = (unsigned int *)calloc(PIXEL_SIZE * PIXEL_SIZE, sizeof(unsigned int));
			printf("OpenMP thread %d on Device %d; Iteration %d:%d \n", thread_No, device_No, iteration_No, ITERATION);
			dim3 bdim(TILE_SIZE, TILE_SIZE);
			dim3 gdim(GRID_SIZE, GRID_SIZE); 
			glensing<<<gdim, bdim>>>(d_lens_x, d_lens_y, d_lens_mass, nobjects, d_results, d_const_struct, globalState, time(NULL) + rand());
			hipMemcpy(temp_results, d_results, PIXEL_SIZE*PIXEL_SIZE*sizeof(unsigned int), hipMemcpyDeviceToHost); 

			for(int k = 0; k<PIXEL_SIZE*PIXEL_SIZE; ++k){
				results[thread_No][k] += temp_results[k];
			}
			int total_x = total_r(temp_results, PIXEL_SIZE * PIXEL_SIZE);
			printf("OpenMP thread %d on Device %d; Iteration %d:%d Total Lenses observed %d\n", device_No, thread_No, iteration_No, ITERATION, total_x);
			free(temp_results);
			hipFree(d_results);
		}
		
		
		/*free the variables*/
		hipFree(d_lens_x);	  
		hipFree(d_lens_y);
		hipFree(d_lens_mass);
		hipFree(d_const_struct);
		hipFree(globalState);
	}	

	int r_c=0, t;
	for(r_c = 0; r_c < PIXEL_SIZE*PIXEL_SIZE; ++r_c){
		for(t=0; t<KERNEL_CALL_NUM; ++t)
			final_result[r_c] += results[t][r_c];
	}
	int total = total_r(final_result, PIXEL_SIZE * PIXEL_SIZE);
	printf("The total num of rays is %d \n", total);

	int highest_c = highest(final_result, PIXEL_SIZE * PIXEL_SIZE);
	write_pgm(final_result, PIXEL_SIZE, PIXEL_SIZE, highest_c);
  
	/*free variables*/
	free(lens_x);
	free(lens_y);
	free(lens_mass);
	free(const_struct);
	free(final_result);
	for(i=0; i<KERNEL_CALL_NUM; ++i)
		free(results[i]);

	hipDeviceReset();
	/*exit the program*/
	return 0;

}